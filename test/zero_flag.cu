#include <hip/hip_runtime.h>

#include <hiprand/hiprand.h>
#include <cassert>
#include <cstdlib>
#include <cstring>

#include <algorithm>
#include <chrono>
#include <fstream>
#include <iostream>
#include <numeric>
#include <random>
#include <string>
#include <type_traits>
#include <unordered_map>
#include <unordered_set>

#define WORKLOAD_PERTHREAD 102400
#define CONSTANT_SIZE 10240

#define checkKernelErrors(expr) do {                                                        \
    expr;                                                                                   \
                                                                                            \
    hipError_t __err = hipGetLastError();                                                 \
    if (__err != hipSuccess) {                                                             \
        printf("File %s: Line %d: '%s' failed: %s\n", __FILE__, __LINE__, # expr, hipGetErrorString(__err));  \
        abort();                                                                            \
    }                                                                                       \
} while(0)

#define CURAND_CALL(x) do { if((x)!=HIPRAND_STATUS_SUCCESS) { \
    printf("Error: %s at %s:%d\n", curandGetStatusString(x),__FILE__,__LINE__);\
    abort();}} while(0)




const char* curandGetStatusString(hiprandStatus_t status) {
// detail info come from http://docs.nvidia.com/cuda/curand/group__HOST.html
    switch(status) {
        case HIPRAND_STATUS_SUCCESS:                     return "CURAND_STATUS_SUCCESS";
        case HIPRAND_STATUS_VERSION_MISMATCH:            return "CURAND_STATUS_VERSION_MISMATCH";
        case HIPRAND_STATUS_NOT_INITIALIZED:             return "CURAND_STATUS_NOT_INITIALIZED";
        case HIPRAND_STATUS_ALLOCATION_FAILED:           return "CURAND_STATUS_ALLOCATION_FAILED";
        case HIPRAND_STATUS_TYPE_ERROR:                  return "CURAND_STATUS_TYPE_ERROR";
        case HIPRAND_STATUS_OUT_OF_RANGE:                return "CURAND_STATUS_OUT_OF_RANGE";
        case HIPRAND_STATUS_LENGTH_NOT_MULTIPLE:         return "CURAND_STATUS_LENGTH_NOT_MULTIPLE";
        case HIPRAND_STATUS_DOUBLE_PRECISION_REQUIRED:   return "CURAND_STATUS_DOUBLE_PRECISION_REQUIRED";
        case HIPRAND_STATUS_LAUNCH_FAILURE:              return "CURAND_STATUS_LAUNCH_FAILURE";
        case HIPRAND_STATUS_PREEXISTING_FAILURE:         return "CURAND_STATUS_PREEXISTING_FAILURE";
        case HIPRAND_STATUS_INITIALIZATION_FAILED:       return "CURAND_STATUS_INITIALIZATION_FAILED";
        case HIPRAND_STATUS_ARCH_MISMATCH:               return "CURAND_STATUS_ARCH_MISMATCH";
        case HIPRAND_STATUS_INTERNAL_ERROR:              return "CURAND_STATUS_INTERNAL_ERROR";
    }
    return "CURAND_STATUS_UNKNOWN_ERROR";
}
    using T = unsigned int;

__global__ void test(T *table, size_t sz) {
    if (threadIdx.x == 0) {
        for (size_t i = 0; i < sz; ++i) {
            // table[i] = i;
            atomicAdd(table + i, 1);
            printf("%u\n", table[i]);
        }
    }
}

int main(int argc, char const *argv[])
{
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    printf("cuda flag: %d\n", int(prop.canMapHostMemory));
        if (!prop.canMapHostMemory) 
            exit(0);
    hipSetDeviceFlags(hipDeviceMapHost);
    hipDeviceSynchronize();

    size_t sz = 1024;
    T *host_table = nullptr;
    T *table = nullptr;

    size_t mem_sz = 1610610944;

    checkKernelErrors(hipMallocManaged(&table, mem_sz));
    // checkKernelErrors(cudaHostAlloc(&host_table, mem_sz, cudaHostAllocMapped));
    // checkKernelErrors(cudaHostGetDevicePointer(&table, host_table, 0));
    // checkKernelErrors(cudaMemset(table, 0, mem_sz));
    test<<<1, 32>>>(table, sz);
    hipDeviceSynchronize();
}
