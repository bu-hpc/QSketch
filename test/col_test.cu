#include <hip/hip_runtime.h>

#include <iostream>
#include <vector>
#include <random>
#include <algorithm>

#define checkKernelErrors(expr) do {                                                        \
    expr;                                                                                   \
                                                                                            \
    hipError_t __err = hipGetLastError();                                                 \
    if (__err != hipSuccess) {                                                             \
        printf("Line %d: '%s' failed: %s\n", __LINE__, # expr, hipGetErrorString(__err));  \
        abort();                                                                            \
    }                                                                                       \
} while(0)

using T = unsigned int;
using C = unsigned int;

using ull = unsigned long long;

const size_t s_sz = 2;

const size_t nblocks = 65536;
const size_t threads_per_block = 32;
const size_t nthreads = nblocks * threads_per_block;

__device__ T hash(T *seed, T s_sz, const T &key) {
    T hv = key;
    for (size_t i = 0; i < s_sz; ++i)
    {
        hv = hv * seed[i];
    }
    return hv;
}


__global__ void col(T *count, T sz, T *seed, T s_sz, T ptm, ull *c) {
    size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    size_t e = (tid + 1) * ptm;
    atomicAdd(c, ull(1));
    for (size_t i = tid * ptm; i < e; ++i)
    {
        // atomicAdd(c, ull(1));
        atomicAdd(&count[hash(seed, s_sz, i) % sz], 1);
        // ++count[i % sz];
    }

    // if (threadIdx.x == 0) {
    //     printf("%u:%u\n", blockIdx.x, count[blockIdx.x]);
    // }
}


int main(int argc, char const *argv[])
{
    size_t tm = std::numeric_limits<T>::max();
    size_t sz = 65536;
    size_t ptm = (tm + 1) / nthreads;

    std::cout << "tm: \t" << tm << std::endl;
    std::cout << "ptm: \t" << ptm << std::endl;
    T *seed;
    checkKernelErrors(hipMalloc((void **)&seed, sizeof(T) * s_sz));
    {
        
        
        T *ts = new T[s_sz];
        std::default_random_engine gen;
        std::uniform_int_distribution<T> dis(1, std::numeric_limits<T>::max());

        for (int i = 0; i < s_sz; ++i)
        {
            ts[i] = (dis(gen));
            std::cout << ts[i] << ",";
        }
        std::cout << std::endl;

        checkKernelErrors(hipMemcpy(seed, ts, s_sz * sizeof(T), hipMemcpyHostToDevice));

        delete [] ts;
    }

    ull *c;
    checkKernelErrors(hipMalloc((void **)&c, sizeof(ull)));
    checkKernelErrors(hipMemset(c, 0, sizeof(ull)));


    T *count = (T *)malloc(sizeof(T) * sz);
    if (count == nullptr) {
        std::cout << "error" << std::endl;
    }
    T *d_count;
    checkKernelErrors(hipMalloc((void **)&d_count, sizeof(T) * sz));
    checkKernelErrors(hipMemset(d_count, 0, sizeof(T) * sz));

    checkKernelErrors((col<<<nblocks, threads_per_block>>>(d_count, sz, seed, s_sz, ptm, c)));
    // checkKernelErrors((test<<<32, 32>>>(d_count)));


    checkKernelErrors(hipMemcpy(count, d_count, sizeof(T) * sz, hipMemcpyDeviceToHost));


    {
        size_t total = 0;
        T max = 0;
        for (size_t i = 0; i < sz; ++i) {
            total += count[i];
            max = std::max(max, count[i]);
        }
        std::cout << "total: \t" << total << std::endl;
        std::cout << "max: \t" << max << std::endl;

        ull hc;
        checkKernelErrors(hipMemcpy(&hc, c, sizeof(ull), hipMemcpyDeviceToHost));
        std::cout << "hc: \t" << hc << std::endl;

        // std::cout << "ave: \t" << 
    }

    delete [] count;
    hipFree(seed);
    hipFree(d_count);

    return 0;
}