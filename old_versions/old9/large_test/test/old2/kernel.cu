
#include <cassert>
#include <cstdlib>
#include <cstring>

#include <algorithm>
#include <chrono>
#include <fstream>
#include <iostream>
#include <numeric>
#include <random>
#include <string>
#include <type_traits>
#include <unordered_map>
#include <unordered_set>

#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>

__global__ void test(int *val) {
    if (threadIdx.x == 0) {
        printf("i\n");
        atomicAdd(val, 1);
    }
}

int main(int argc, char const *argv[])
{

    size_t fifo_sz;
    hipDeviceGetLimit(&fifo_sz,hipLimitPrintfFifoSize);
    // std::cout << fifo_sz << std::endl;
    hipDeviceSetLimit(hipLimitPrintfFifoSize, fifo_sz * 8);

    int nb = atoi(argv[1]); 
    int nt = atoi(argv[2]);
    int hval;
    int *dval;
    hipMalloc(&dval, sizeof(int));
    hipMemset(dval, 0, sizeof(int));
    test<<<nb, nt>>>(dval);
    hipMemcpy(&hval, dval, sizeof(int), hipMemcpyDeviceToHost);
    // std::cout << hval << std::endl;
    // while (1) {
    //     std::cerr << "l" << std::endl;
    // }

    hipDeviceSynchronize();
    hipDeviceReset();
    return 0;
}