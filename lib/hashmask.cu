#include "hip/hip_runtime.h"
#include <qsketch.h>
#include <bitset>
namespace qsketch {

bool set(uchar *hash_mask, uint id) {
    uint cid = id / bits<uchar>;
    uchar b = 1 << (id % bits<uchar>);

    // std::cout << id << ":" << std::bitset<8>(b)
        // std::bitset<8>(hash_mask[cid]) 
        // << std::endl;
    // std::cout << std::bitset<8>(hash_mask[cid] & b) << std::endl;
    if ((hash_mask[cid] & b) == 0) {
        hash_mask[cid] |= b;
        // std::cout << std::bitset<8>(hash_mask[cid]) << std::endl;
        return true;
    }

    return false;
}

bool get(uchar *hash_mask, uint id) {
    uint cid = id / bits<uchar>;
    uchar b = 1 << (id % bits<uchar>);
    return hash_mask[cid] & b;
}

__device__ __host__ inline bool get_hashmask(uchar hash_mask, uint id) {
    return false;
}

uchar *generate_hashmask(uchar *hash_mask_table, size_t n, size_t m, 
    size_t hash_mask_ones, size_t padding,
    uint **index_hash_mask_table_ptr) {
    /*
        n : the number of hash_mask
        m : the size of hash_mask, (bits)
        padding: the last bits which are zero
    */
    #ifdef QSKETCH_DEBUG
        std::cout << "generate_hashmask" << std::endl;
        std::cout << n << ", " << m << ", " << hash_mask_ones << ", " << padding << std::endl; 
    #endif
    size_t hash_mask_sz = ceil<size_t>(m, bits<uchar>);

    // std::cout << hash_mask_sz << std::endl;

    uint *index_hash_mask_table = nullptr;
    if (hash_mask_table == nullptr) {
        // size_t mem_sz = ceil<size_t>(n * m, sizeof(uchar));
        size_t mem_sz = n * hash_mask_sz;
        hash_mask_table = cpu_tool<uchar>.zero(hash_mask_table, mem_sz);
        if (index_hash_mask_table_ptr != nullptr
            && *index_hash_mask_table_ptr == nullptr) {
            index_hash_mask_table = *index_hash_mask_table_ptr = cpu_tool<uint>.zero(nullptr, n * hash_mask_ones);
        }
    }

    auto &eng = cpu_tool<uint>.eng; 
    size_t lm = m;
    std::uniform_int_distribution<uint> dis(0, lm - padding - 1);

    for (size_t i = 0; i < n; ++i) {
        size_t j = 0;
        while (j < hash_mask_ones) {
            uint id = dis(eng);
            if (set(hash_mask_table + i * hash_mask_sz, id)) {
                if (index_hash_mask_table_ptr != nullptr) {
                    // *((*index_hash_mask_table_ptr) + i * hash_mask_ones + j) = id;
                    index_hash_mask_table[i * hash_mask_ones + j] = id;
                    // std::cout << 
                }
                j++;

                // std::cout << id << ",";
            }
        }
        // std::cout << std::endl;
    }

    return hash_mask_table;
}


}